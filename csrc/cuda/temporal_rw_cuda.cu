#include "hip/hip_runtime.h"
#include "temporal_rw_cuda.h"

#include <ATen/cuda/HIPContext.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "utils.cuh"

#define THREADS 1024
#define BLOCKS(N) (N + THREADS - 1) / THREADS

__device__ int64_t binary_search_min_cuda(const int64_t tgt,
                                 const int64_t st_idx,
                                 const int64_t en_idx,
                                 const int64_t *ts) {
  /*
    Return lowest idx of ts that is >= tgt
    Assumes ts is sorted between ts[start] and ts[end]

    Rewritten as a loop instead of recursion for performance improvement
  */
  // Convert from const
  int st = st_idx;
  int en = en_idx;

  while (en-st > 1) {
    auto len = en-st;
    auto half = len >> 1;
    auto val_at_half = ts[st+half];

    if (val_at_half >= tgt) {
        en = en-half;
    } else {
        st = st+half;
    }
  }

  if (ts[st] >= tgt) {
    return st;
  }
  return en;
}

__device__ int64_t binary_search_max_cuda(const int64_t tgt,
                                          const int64_t st_idx,
                                          const int64_t en_idx,
                                          const int64_t *ts) {
  int st = st_idx;
  int en = en_idx;

  while (en-st > 1) {
    auto len = en-st;
    auto half = len >> 1;
    auto val_at_half = ts[st+half];

    if (val_at_half <= tgt) {
        st = st+half;
    } else {
        en = en-half;
    }
  }

  if (ts[en] <= tgt) {
    return en;
  }
  return st;
}

__global__ void uniform_sampling_kernel(const int64_t *rowptr,
                                        const int64_t *col,
                                        const int64_t *ts,
                                        const int64_t *start, const float *rand,
                                        int64_t *n_out, int64_t *e_out,
                                        const int64_t walk_length,
                                        const int64_t t_start,
                                        const int64_t t_end,
                                        const bool reverse,
                                        const int64_t numel) {

  const int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (thread_idx < numel) {
    int64_t n_cur = start[thread_idx], e_cur = 0, row_start, row_end, rnd;

    n_out[thread_idx] = n_cur;

    int64_t t = (reverse) ? t_end : t_start;

    for (int64_t l = 0; l < walk_length; l++) {
      row_start = rowptr[n_cur], row_end = rowptr[n_cur + 1];

      // Skip search if we already know we're at a terminal node/time
      if (e_cur == -1) {
        row_start = row_end;
      } else {
        // When traversing backward keep row_start the same, and decrease row_end
        if (reverse) {
          row_start = (t_start) ? binary_search_min_cuda(t_start, row_start, row_end, ts) : row_start;
          row_end = binary_search_max_cuda(t, row_start, row_end, ts);
        // Else, keep row_end the same and increase row_start
        } else {
          row_start = binary_search_min_cuda(t, row_start, row_end, ts);
          row_end = (t_end) ? binary_search_max_cuda(t_end, row_start, row_end, ts) : row_end;
        }
      }

      if (row_end - row_start == 0) {
        e_cur = -1;
      } else {
        rnd = int64_t(rand[l * numel + thread_idx] * (row_end - row_start));
        e_cur = row_start + rnd;
        n_cur = col[e_cur];
        t = ts[e_cur];
      }
      n_out[(l + 1) * numel + thread_idx] = n_cur;
      e_out[l * numel + thread_idx] = e_cur;
    }
  }
}


std::tuple<torch::Tensor, torch::Tensor>
temporal_random_walk_cuda(torch::Tensor rowptr, torch::Tensor col, torch::Tensor ts, torch::Tensor start,
                 int64_t walk_length, int64_t t_start, int64_t t_end, bool reverse) {
  CHECK_CUDA(rowptr);
  CHECK_CUDA(col);
  CHECK_CUDA(start);
  CHECK_CUDA(ts);
  c10::cuda::MaybeSetDevice(rowptr.get_device());

  CHECK_INPUT(rowptr.dim() == 1);
  CHECK_INPUT(col.dim() == 1);
  CHECK_INPUT(start.dim() == 1);
  CHECK_INPUT(ts.dim() == 1);

  auto n_out = torch::empty({walk_length + 1, start.size(0)}, start.options());
  auto e_out = torch::empty({walk_length, start.size(0)}, start.options());

  auto stream = at::cuda::getCurrentCUDAStream();

  auto rand = torch::rand({start.size(0), walk_length},
                            start.options().dtype(torch::kFloat));

  uniform_sampling_kernel<<<BLOCKS(start.numel()), THREADS, 0, stream>>>(
      rowptr.data_ptr<int64_t>(), col.data_ptr<int64_t>(), ts.data_ptr<int64_t>(),
      start.data_ptr<int64_t>(), rand.data_ptr<float>(),
      n_out.data_ptr<int64_t>(), e_out.data_ptr<int64_t>(),
      walk_length, t_start, t_end, reverse,
      start.numel());

  return std::make_tuple(n_out.t().contiguous(), e_out.t().contiguous());
}
